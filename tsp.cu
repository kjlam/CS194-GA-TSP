#include "hip/hip_runtime.h"
/*
 * pet
 * TSP.cc
 *
 * Created on: Apr 17, 2012
 * Author: Kelvin
 */
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include "tsp.h"
#include <fstream>
#include <iostream>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

using std::vector;
using namespace std;

int num_cities = 1;
int population_size = 0;
int greedy_selection_percentage;
int num_closer_way_points;
int group_size;
int mutation_percentage;
int termination_step;
//int** distance_matrix;
float* distance_matrix;
float* d_distance_matrix;
int ** closest_neighbors;
tour* population; //Tour new_population[]

tour *h_children;
tour *h_parent_set1;
tour *h_parent_set2;
tour* h_index_of_parent_set1;
tour* h_index_of_parent_set2;
tour* h_cycles;

tour* d_children;
tour* d_parent_set1;
tour* d_parent_set2;
tour* d_index_of_parent_set1;
tour* d_index_of_parent_set2;
tour* d_cycles;
tour* d_population;

thrust::device_vector<tour> thrust_population; 
thrust::device_vector<tour> thrust_children;


double
timestamp (){
	struct timeval tv;
	gettimeofday (&tv, 0);
	return tv.tv_sec + 1e-6*tv.tv_usec;
}



/*
 * partition: used in quick_select
 */
int array_partition(float* input, int p, int r)
{
	float pivot = input[r];
	
	while ( p < r )
	{
		while ( input[p] < pivot )
			p++;
		
		while ( input[r] > pivot )
			r--;
		
		if ( input[p] == input[r] )
			p++;
		else if ( p < r ) {
			int tmp = input[p];
			input[p] = input[r];
			input[r] = tmp;
		}
	}
	
	return r;
}

/*
 * quickselect: finds the kth smallest value within the index of p and r of input
 */
int array_quick_select(float* input, int p, int r, int k)
{
	while(1){
		if ( p == r )
			return input[p];
		int j = array_partition(input, p, r);
		int length = j - p + 1;
		if ( length == k )
			return input[j];
		else if ( k < length )
			r = j-1;
		else{
			k -= length;
			p = j + 1;
		}
	}
}

/*
 * vector input version of partition
 */
int vector_partition(vector<int> input, int p, int r){
	float pivot = input[r];
	
	while ( p < r )
	{
		while ( input[p] < pivot )
			p++;
		
		while ( input[r] > pivot )
			r--;
		
		if ( input[p] == input[r] )
			p++;
		else if ( p < r ) {
			int tmp = input[p];
			input[p] = input[r];
			input[r] = tmp;
		}
	}
	
	return r;
}


/*
 * vector version of quickselect
 */
int vector_quick_select(vector<int> input, int p, int r, int k)
{
	while(1){
		if ( p == r )
			return input[p];
		int j = vector_partition(input, p, r);
		int length = j - p + 1;
		if ( length == k )
			return input[j];
		else if ( k < length )
			r = j-1;
		else{
			k -= length;
			p = j + 1;
		}
	}
}


/*
 * generates a distance matrix in distance_matrix
 */
void generate_distance_matrix(){
	//TODO
	
}
/*
 * void generate_initial_population(): generates population by
 * calling generate tour many times,
 */
void generate_initial_population(){
	int *linear_cities = new int[num_cities];
	for(int i = 0; i < num_cities; i++){
		linear_cities[i] = i;
	}
	for(int j =0; j < population_size; j++){
		generate_tour(linear_cities, j);
		/*
		 *	cout << "tour number " << j << ": " << endl;
		 *	for (int k = 0; k < num_cities; k++) {
		 *	cout << population[j].tour[k] << " ";
		 }
		 cout << endl;
		 */
	}
}

/*
 * DEPRECATED find_n_closest_neighbors finds the n closest neighbors for all the cities
 * and place them in closest_neighbors 2d array NOT WORKING AS INTENDED, QUICK_SELECT 
 * RETURNS VALUE NOT INDEX NEED TO FIND INDEX VALUE
 *
 * void generate_closest_neighbors(){
 *	for(int i = 0; i < population_size; i++){
 *		int closest_index = 0;
 *		//num_closer_way_points + 1 used as quick_select will pick out i as well, since distance is 0 with itself
 *		float m = array_quick_select(distance_matrix[i], 0, population_size - 1, num_closer_way_points + 1);
 *		
 *		closest_neighbors[i][closest_index] = m;
 *		closest_index++;
 *		for(int k = 0; k < population_size; k++){
 *			if(distance_matrix[i][k] < m && distance_matrix[i][k] != 0 ){
 *				closest_neighbors[i][closest_index] = distance_matrix[i][k];
 *				closest_index++;
 *				if(closest_index == num_closer_way_points){
 *					return;
 }
 }
 }
 }
 }
 */


/*
 * Tour generate_tour(): returns a tour based on the distance_matrix
 * paper had 2 methods for picking the next waypoint in a tour that
 * were influenced by the greedy_selection_percentage
 */
void generate_tour(int* linear_cities, int index){
	tour new_tour;
	new_tour.fitness = 0;
	//new_tour.tour = new int[num_cities];
	//new_tour.tour_lengths = new float[num_cities];
	//cout << "tour number: " << index << endl;
	//vector of available_cities, cities will get deleted from this resizable array as they get added to the tour
	//vector<int> available_cities(linear_cities, linear_cities + sizeof(linear_cities)/sizeof(int));
	vector<int> available_cities(linear_cities, linear_cities + num_cities);
	
	//first city in tour will always be the first city (doesn't matter where we start as tours will loop through all cities
	
	new_tour.path[0] = 0;
	int current_city = 0;
	available_cities.erase(available_cities.begin());
	int next_city = 0;
	
	//loop num_cities times to form the tour
	for(int i = 1; i < num_cities; i++){
		int selection = (rand() % 100) + 1;
		//cout << "213 selection percent " << selection << endl;
		//choose greedily
		//if greedy strategy selected and the number of cities available is greater than the number of closer way points
		//perform the greedy strategy
		if((selection < greedy_selection_percentage) and (available_cities.size() > num_closer_way_points)){
			//cout << "217 greedy selection" << endl;
			//TODO: check that the ranodm selection producing desired values
			int random_closest= rand() % num_closer_way_points;
			next_city = vector_quick_select(available_cities, 0, available_cities.size() - 1, random_closest);
		}else // choose next city randomly
		{
			//cout << "224 city chosen randomly" << endl;
			int city_index = rand() % available_cities.size();
			//cout<< "227 next city " << city_index << endl;
			next_city = available_cities[city_index];
		}
		//loop through the available_cities vector until u found the city value selected to be added
		//delete the value from the available_cities vector
		for(int j = 0; j < available_cities.size(); j++){
			if(available_cities[j] == next_city){
				available_cities.erase(available_cities.begin() + j);
				break;            
			}
		}
		
		// add the city to the tour, calculate the fitness it adds, and add the tour_length as well
		
		//cout << next_city << endl;
		new_tour.path[i] = next_city;
		//new_tour.tour_lengths[i-1] = distance_matrix[current_city*num_cities + next_city];
		new_tour.fitness += distance_matrix[current_city * num_cities + next_city];
		current_city = next_city;
	}
	
	//compute the final fitness and tour_length connecting the final city to the first city
	//new_tour.tour_lengths[num_cities-1] = distance_matrix[current_city * num_cities + 0];
	new_tour.fitness += distance_matrix[current_city * num_cities + next_city];
	population[index] = new_tour;
}

/*
 * DEPRECATED, method's funcionality built into generate_tour
 * int greedy_selection(int city): greedy method for selecting next
 * waypoint in tour in generateTour
 */
int greedy_selection(int city){
	return 0;
}

/*
 * DEPRECATED, method's functionality build into generate_tour
 * int random_selection(int city): randomly pick 2 waypoints among
 * list of cities (not including the city entered into the method,
 * then pick the shorter cost of the two
 */
int random_selection(int city){
	return 0;
	
}


/*
 * DEPRECATED, just uses quicksort
 * select_group(int group_size): quicksorts the population
 * group_size of the most optimal tours (run sort_population first
 * to figure out best tours)
 */
void select_group(int group_size){
	
}


/*//given a tour with initialized tour array, returns fitness of tour
float compute_fitness(tour t){
	float fitness = 0;
	for(int i = 0; i < num_cities -1; i ++){
		//cout << "305 " << t.path[i] << " " << t.path[i+1];
		fitness += distance_matrix[t.path[i] * num_cities + t.path[i+1]];
		if(t.path[i] != -1 and t.path[i+1] != -1){
		//	cout << "299 " <<  t.path[i]  << " " << t.path[i+1] << " " << endl;
		}
	}
	fitness += distance_matrix[t.path[num_cities-1] * num_cities + t.path[0]];
	return fitness;
}
*/

__global__ static void
	compute_fitness(tour* children, float* d_matrix, int num_cities){
		int id = blockIdx.x*blockDim.x + threadIdx.x;
		children[id].fitness = 0;
		for(int i = 0; i < num_cities - 1; i++){
			children[id].fitness += d_matrix[ children[id].path[i] * num_cities + children[id].path[i+1]];
		}
		children[id].fitness += d_matrix[ children[id].path[num_cities - 1] * num_cities + children[id].path[0]];
	}


 


/*
 * void sort_population(): sorts population so that the GA can
 * organize (quicksort)
 */
void sort_population(){
	qsort_population(0, population_size - 1, population);
}

void qsort_population(int left, int right, tour* population) {
	if (right > left) {
		int pivotIndex = rand() % (right - left + 1);
		tour pivot = population[left + pivotIndex];
		int pivotfitness = pivot.fitness;
		population[left + pivotIndex] = population[right];
		population[right] = pivot;
		
		int i = left - 1;
		int j = right;
		
		do {
			do { i++; } while (population[i].fitness < pivotfitness);
			do { j--; } while (population[j].fitness > pivotfitness && j > left);
			if (i < j) {
				tour ith = population[i];
				population[i] = population[j];
				population[j] = ith;
			}
			
		} while (i < j);
		
		population[right] = population[i];
		population[i] = pivot;
		qsort_population(left, i - 1, population);
		qsort_population(i + 1, right, population);
	}
}



struct tour_pair{
	tour t1; 
	tour t2;
};


/*
 * Tour crossover(Tour parent1, Tour Parent2): Crossover of
 * 2 parents and then compute the length
 */
__global__ static void
	crossover(tour* parent1, tour* parent2, tour* children, tour* indexOfParent1, tour* indexOfParent2, tour* cycles, int num_cities, int group_size){
		int instance = blockIdx.x*blockDim.x + threadIdx.x;
		if(instance >= group_size/2){
			return;
		}

		for (int k = 0; k < num_cities; k++) {
			//children[2*instance].path[k] = -1;
			//children[2*instance + 1].path[k] = -1;
			cycles[instance].path[k] = -1;
		}
		
		int cycle_index = 0;
		
		int p1Index = parent1[instance].path[0];
		cycles[instance].path[cycle_index] = p1Index;
		cycle_index = cycle_index + 1;

		for(int i = 0; i < num_cities; i ++){
			int p1_value_at_i = parent1[instance].path[i];
			int p2_value_at_i = parent2[instance].path[i];
			indexOfParent1[instance].path[p1_value_at_i] = i;
			indexOfParent2[instance].path[p2_value_at_i] = i;
			children[2*instance].path[i] = p1_value_at_i;
			children[2*instance + 1].path[i] = p2_value_at_i;
		}
		
	
		
		children[2*instance].fitness = 0;
		children[2*instance + 1].fitness = 0;
		
		//int p2city = indexOfParent2[instance].path[p1Index];
		int p2city = children[2*instance +1].path[p1Index];
		 p1Index = indexOfParent1[instance].path[p2city];
		
		while(p1Index != cycles[instance].path[0]){
			cycles[instance].path[cycle_index] = p1Index;
			cycle_index++;
			p2city = children[2*instance+1].path[p1Index];
			p1Index = indexOfParent1[instance].path[p2city];
		}
		
		for(int i = 0; i < num_cities; i++){
			if(cycles[instance].path[i] != -1){
				int reverse = cycles[instance].path[i];
				int tmp = children[2*instance].path[reverse];
				children[2*instance].path[reverse] = children[2*instance+1].path[reverse];
				children[2*instance+1].path[reverse] = tmp;
			}
		}
			

		/*
		while(1) {
			bool visited = false;
			for (int j = 0; j < num_cities; j++) {
				if (children[2*instance].path[j] == p2city) {
					visited = true;
				}
			}
			if (visited) {
				break;
			}
			
			//since p2city hasn't yet been visited by child1, find where p2city occurs in parent1 and insert it into child1 at the same index
			for (int j = 0; j < num_cities; j++) {
				if (parent1[instance].path[j] == p2city) {
					children[2*instance].path[j] = p2city;
					children[2*instance + 1].path[j] = parent2[instance].path[j];
					p2city = parent2[instance].path[j];
				}
			}
		}
		
		//fill in the -1 values in the children
		for (int k = 0; k < num_cities; k++) {
			if (children[2*instance].path[k] == -1) {
				children[2*instance].path[k] = parent2[instance].path[k];
				children[2*instance + 1].path[k] = parent1[instance].path[k];
			}
		}
		//can’t compute fitness here as requires distance_matrix
		//child1.fitness = compute_fitness(child1);
		//child2.fitness = compute_fitness(child2);
		//children[index] = child1;
		//children[index + 1] = child2;
		*/
}
	


/*
 * parallalel mutate 
 */

__global__ void parallel_mutate(tour* d_children, int* d_mutate_indices, int group_size, int num_cities){

        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int tid = threadIdx.x;
       
        //tour tours[32];
        //tours[tid] = d_children[i];
       
        int num1 = d_mutate_indices[2*blockIdx.x] % num_cities;
        int num2 = d_mutate_indices[2*blockIdx.x+1] % num_cities;
		
		int start = min(num1, num2);
		int end = max(num1, num2);

        if (i < group_size) {
                
                while (start < end) {
                        int temp = d_children[i].path[start];
                        d_children[i].path[start] = d_children[i].path[end];
                        d_children[i].path[end] = temp;
                        start++;
                        end--;
                }
                
               /* while (start < end) {
                        int temp = tours[tid].path[start];
                        tours[tid].path[start] = tours[tid].path[end];
                        tours[tid].path[end] = temp;
                        start++;
                        end--;
                }
               */
               // d_children[i] = tours[tid];
        }
}

__global__ static void 
	select_group(tour* d_p, tour* d_p1, tour* d_p2){
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		
		d_p1[id]= d_p[id*2];
		d_p2[id] = d_p[id*2 + 1];
		
	}


void create_children(){
	/*cout << "population before sorting\n";
	for (int i = 0; i < population_size; i++) {
		cout << population[i].fitness << " ";
	}
	cout << endl;
	*/
	//qsort population (parallelize?)
	//qsort_population(0, num_cities - 1, population);
	/*thrust::host_vector <tour> h_population(population_size);
	for(int k = 0; k < population_size; k++){
		h_population[k] = population[k];
	}
	thrust::device_vector<tour> d_population = h_population;
	thrust::sort(d_population.begin(), d_population.end());
	thrust::copy(d_population.begin(), d_population.end(), h_population.begin());
	*/
	
	thrust::sort(thrust_population.begin(), thrust_population.end());
	
	/*
	for(int m = 0; m < population_size; m++){
		population[m] = h_population[m];
	}
	*/
	
	int num_threads = group_size/2;
	int num_blocks = (num_threads + 31)/32;
	dim3 block(32, 1);
	dim3 grid(num_blocks, 1);
	
	select_group<<<grid, block>>>(d_population, d_parent_set1, d_parent_set2);
	//cout << "before crossover" << endl;
	//apply crossover on adjacent pairs of elements in the parent set
	//thrust::transform(d_parent_set1.begin(), d_parent_set1.end(), d_parent_set2.begin(), d_children.begin(), crossover_functor(num_cities));
	crossover<<<grid, block>>>(d_parent_set1, d_parent_set2, d_children, d_index_of_parent_set1, d_index_of_parent_set2, d_cycles, num_cities, group_size);
	
	
	num_threads = group_size;
	num_blocks = (num_threads + 31)/32;
	dim3 block1(32, 1);
	dim3 grid1(num_blocks, 1);
	int* h_random_arr = new int[num_blocks*2];
	for(int i = 0; i < num_blocks * 2; i++){
		h_random_arr[i] = rand();
	}
	
	int* d_random_arr = 0;
	
	hipMalloc((void**)&d_random_arr, sizeof(int) * num_blocks * 2);
	hipMemcpy(d_random_arr, h_random_arr, sizeof(int) * num_blocks * 2, hipMemcpyHostToDevice);
	
	parallel_mutate<<<grid1, block1>>> (d_children, d_random_arr, group_size, num_cities);
	
	//cout << "after crossover" << endl;
	//hipMemcpy(h_children, d_children, sizeof(tour)*group_size, hipMemcpyDeviceToHost);
	/*
	//hipFree(d_children);
	hipFree(d_parent_set1);
	hipFree(d_parent_set2);
	hipFree(d_index_of_parent_set1);
	hipFree(d_index_of_parent_set2);
	hipFree(d_cycles);
	*/
	
	//cout << "525 after cudaFrees" << endl;
	//serial mutation
	/*
	 for (int i = 0; i < group_size; i ++){
	 		
	 		int mutate_or_not = rand() % 100 + 1;
	 		if(mutate_or_not < mutation_percentage){
	 			mutate(&h_children[i]);
		}
		//h_children[i].fitness = compute_fitness(children[i]);
	 }
	 */

	//cout << "535 after mutaion" << endl;
	//tour* c = new tour[group_size];
	//thrust::host_vector<tour_pair> h_children(group_size);
	//thrust::transform(d_children.begin(), d_children.end(), d_children.begin(), mutate_functor());
	
	//cout << "478 " << h_children[0].path[3] << endl;
	//cout << "478 " << h_children[0].path[40] << endl;
	//thrust::copy(d_children.begin(), d_children.end(), h_children.begin());


	

	//hipMemcpy(d_children, h_children, sizeof(tour)*group_size, hipMemcpyHostToDevice);
	
	compute_fitness<<<grid1, block1>>>(d_children, d_distance_matrix, num_cities);
	/*for (int i = 0; i < group_size; i++){
		//cout << "479 " << h_children[i].fitness << endl;
		//cout << "before h_children[i]" << endl;
		h_children[i].fitness = compute_fitness(h_children[i]);
		//cout << "child fitness : " << i << " " << h_children[i].fitness << endl;
	}*/
	
	hipFree(d_random_arr);
	
	/*hipMemcpy(h_children, d_children, sizeof(tour)*group_size, hipMemcpyDeviceToHost);
	hipFree(d_children);
		//hipFree(d_children);
	hipFree(d_parent_set1);
	hipFree(d_parent_set2);
	hipFree(d_index_of_parent_set1);
	hipFree(d_index_of_parent_set2);
	hipFree(d_cycles);
	
	//cout << "546 after fitness computation" << endl;
	
	return h_children;
	*/
}



/*
 * Tour mutate(Tour t ): mutates the tour and returns the
 * optimal tour of the mutated or original
 */

void mutate(tour* t ){
	int start;
	int end;
	do {
		start = rand() % (num_cities -1) + 1;
		end = rand() % (num_cities - 1) + 1;
	} while ((start >= end) or (start == 0 and end == (num_cities - 1)));
	
	while (start < end) {
		int temp = t->path[start];
		t->path[start] = t->path[end];
		t->path[end] = temp;
		start++;
		end--;
	}
}




/*
 * create_new_generation sorts the current population tour array and the children tour array, and then proceeds to replace
 * the group_size weakest tours in the population array with the children if the fitness of the child is higher
 */
__global__ void
	create_new_generation(tour* population, tour* children, int population_size, int group_size){
	//qsort_population(0, population_size -1, population);
	//qsort_population(0, group_size -1, children);
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int population_index = population_size - group_size;
	//int children_index = 0;
	/*
	 * this for loop only selects the group_size best between group_size lowest from previous generation and 
	 * the new children
	 */
	/*for(int i = 0; i < group_size && population_index < population_size; i ++){

		if(children[children_index].fitness < population[population_index].fitness){
			population[population_index] = children[children_index];
			population_index ++;
		}
		children_index++;
	}*/
	//replaces group_size worst solutions with children regardless of fitness
	population[population_index + id] = children[id];
	
	
}



/*
 * Tour run(): generates initial population, then runs through
 * a for loop that goes through the genetic algorithm until the
 * termination_step, return the best population
 */
void run_genetic_algorithm(){
	/*distance_matrix = new int*[num_cities];
	 *	for(int i = 0; i < num_cities; i ++){
	 *	distance_matrix[i] = new int[num_cities];
}
*/
	population = new tour[population_size];
	closest_neighbors = new int*[num_cities];
	for(int i = 0; i < num_cities; i ++){
		closest_neighbors[i] = new int[num_closer_way_points];
		
	}
	
	generate_initial_population();
	
	thrust::host_vector<tour> h_population(population_size);
	for(int i = 0; i < population_size; i++){
		h_population[i] = population[i];
	}
	thrust_population = h_population;
	thrust::sort(thrust_population.begin(), thrust_population.end());
	//move distance_matrix into gpu
	d_distance_matrix = 0;
	
	hipMalloc((void**)&d_distance_matrix, sizeof(float) * num_cities*num_cities);
	
	int num_threads = group_size/2;
	int num_blocks = (num_threads + 31)/32;
	dim3 block(32, 1);
	dim3 grid(num_blocks, 1);
	h_children = new tour[group_size];
	h_parent_set1 = new tour[group_size/2];
	h_parent_set2 = new tour[group_size/2];
	h_index_of_parent_set1 = new tour[group_size/2];
	h_index_of_parent_set2 = new tour[group_size/2];
	h_cycles = new tour[group_size/2];
	
	
	for(int i = 0 ; i < group_size; i =i+2){
		h_parent_set1[i/2] = population[i];
		h_parent_set2[i/2] = population[i+1];
		//cout << "h_parent_set1 " << i/2 << " " <<  h_parent_set1[i/2].fitness << endl;
		//cout << "h_parent_set2 " << i/2 << " " <<h_parent_set2[i/2].fitness << endl;
	} 
	
	d_children = 0;
	d_parent_set1 = 0;
	d_parent_set2 = 0;
	d_index_of_parent_set1 = 0;
	d_index_of_parent_set2 = 0;
	d_cycles = 0;
	d_population = thrust::raw_pointer_cast(thrust_population.data());
	d_children = thrust::raw_pointer_cast(thrust_children.data());
	
	//cout << "497 before cudaMallocs" << endl;
	hipMalloc((void**)&d_children, sizeof(tour)*group_size);
	hipMalloc((void**)&d_parent_set1, sizeof(tour)*group_size/2);
	hipMalloc((void**)&d_parent_set2, sizeof(tour)*group_size/2);
	hipMalloc((void**)&d_index_of_parent_set1, sizeof(tour)*group_size/2);
	hipMalloc((void**)&d_index_of_parent_set2, sizeof(tour)*group_size/2);
	hipMalloc((void**)&d_cycles, sizeof(tour)*group_size/2);
	//hipMalloc((void**)&d_population, sizeof(tour) * population_size);
	//cout << "504 before cudaMemcpys " << endl;
	
	hipMemcpy(d_children, h_children, sizeof(tour)*group_size, hipMemcpyHostToDevice);
	hipMemcpy(d_parent_set1, h_parent_set1, sizeof(tour)*group_size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_parent_set2, h_parent_set2, sizeof(tour)*group_size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_index_of_parent_set1, h_index_of_parent_set1, sizeof(tour)*group_size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_index_of_parent_set2, h_index_of_parent_set2, sizeof(tour)*group_size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_cycles, h_cycles, sizeof(tour)*group_size/2, hipMemcpyHostToDevice);
	hipMemcpy(d_distance_matrix, distance_matrix, sizeof(float)*num_cities*num_cities, hipMemcpyHostToDevice);
	//hipMemcpy(d_population, population, sizeof(tour) * population_size);
	

	num_threads = group_size;
	num_blocks = (num_threads + 31)/32;
	dim3 block1(32, 1);
	dim3 grid1(num_blocks, 1);

	for(int j = 0; j < termination_step; j++){
		create_children();
		thrust::sort(thrust_children.begin(), thrust_children.end());
		create_new_generation<<<block1, grid1>>>(d_population, d_children, population_size, group_size);

	}
	
	/*for(int m = 0; m < population_size; m++){
		population[m] = h_population[m];
	}
	*/
	thrust::sort(thrust_population.begin(), thrust_population.end());
		
	hipFree(d_children);
	hipFree(d_parent_set1);
	hipFree(d_parent_set2);
	hipFree(d_index_of_parent_set1);
	hipFree(d_index_of_parent_set2);
	hipFree(d_cycles);
	hipFree(d_distance_matrix);
	

	
	hipMemcpy(population, d_population, sizeof(tour)*population_size, hipMemcpyDeviceToHost);
	//hipMemcpy(best_tour, d_population[0], sizeof(tour), hipMemcpyDeviceToHost);
	
	hipFree(d_population);
	
	
	cout << "Fitness: " << population[0].fitness << endl << "Tour \n";
	for (int i = 0; i < num_cities; i++){
		cout << population[0].path[i] << endl;
	}
	//print_best_tour();
}

/*
 * print_best_tour: prints out the best tour along with its fitness, run at the end of
 * run_genetic_algorithm()
 */
void print_best_tour(){
	qsort_population(0, population_size - 1, population);
	cout << "Best Tour Generated After " << termination_step << "generations \n";
	cout << "Fitness: " << population[0].fitness << endl << "Tour \n";
	for (int i = 0; i < num_cities; i++){
		cout << population[0].path[i] << endl;
	}
}


/*
 * program requires all these parameters
 * program_name tsp_file num_cities population_size greedy_selection_percentage num_closer_way_points group_size
 * mutation_percentage termination_step
 */
int main(int argc, char** argv){
	if(argc != 9){
		cout << "not enough arguments\n" << " program requires 9 arguments "
		<< "program_name tsp_file num_cities population_size greedy_selection_percentage"
		<< "num_closer_way_points group_size mutation_percentage termination_step";
		return 0;
		
		
		//./a.out ./burma14.tsp 14 100 20 5 20 50 10
		
	}
	/*
	 *	num_cities = 10;
	 *	population_size = 10;
	 *	greedy_selection_percentage = 80;
	 *	num_closer_way_points = 3;
	 *	group_size = 3;
	 *	mutation_percentage = 70;
	 *	termination_step = 20;
	 */
	double start = timestamp();
// your codes here

	
	char* filename = argv[1];
	const char * num_cities_string = argv[2];
	num_cities = atoi(num_cities_string);
	population_size = atoi(argv[3]);
	greedy_selection_percentage = atoi(argv[4]);
	num_closer_way_points = atoi(argv[5]);
	group_size = atoi(argv[6]);
	mutation_percentage = atoi(argv[7]);
	termination_step = atoi(argv[8]);
	distance_matrix = new float[num_cities * num_cities];
	/*for loop no longe rneeded as changed distance_matrix to 1d array
	 * for (int i = 0; i< num_cities; i++) {
	 *		distance_matrix[i] = new float[num_cities];
}
*/
	//initialize the random seed, ONLY CALL ONCE in program 
	srand(time(NULL));
	
	coordinates cities[num_cities];
	
	char line[256];
	ifstream myfile;
	myfile.open(filename);
	
	const char * lastline = "NODE_COORD_SECTION";
	while (1) {
		myfile.getline(line, 256);
		//cout << "asdf\n";
		if (!strcmp(line, lastline)) {
			break;
		}
	}
	
	for (int i = 0; i < num_cities; i++) {
		myfile.getline(line, 256);
		char * pch;
		pch = strtok(line, " ");
		pch = strtok(NULL, " ");
		cities[i].x = atoi(pch);
		pch = strtok(NULL, " ");
		cities[i].y = atoi(pch);
	}
	
	for (int i = 0; i < num_cities; i++) {
		for (int j = 0; j < num_cities; j++) {
			distance_matrix[i * num_cities + j] = sqrt(pow(cities[i].x - cities[j].x,2) + pow(cities[i].y - cities[j].y,2));
		}
	}
	
	myfile.close();
	
	cout << "file parsing reached" << endl;
	run_genetic_algorithm();
	/* no longer needed as distance_matrix is a 1d array
	 *	for (int i = 0; i < num_cities; i++) {
	 *		delete[] distance_matrix[i];
}
*/
	delete[] distance_matrix;
	double end = timestamp();
	cout << "time elapsed" << (start - end) * 1000 << endl;
}
